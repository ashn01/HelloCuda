﻿#include <cstdio>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


__global__ void hello(void) {
	printf("hello CUDA %d!\n", threadIdx.x);
}

int main(void) {
	hello <<< 8, 2 >>> ();
	fflush(stdout);
	return 0;
}